#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
// added math library
#include<math.h>
#include <stdio.h>
// added one set matrix dimension
#define SIZE 3

hipError_t addWithCuda(int c[][SIZE], const int a[][SIZE], const int b[][SIZE], unsigned int size, dim3 threads);

// changed the parameters from 1D vectors to 2D matrices with one dimension set
__global__ void addKernel(int c[][SIZE], const int a[][SIZE], const int b[][SIZE])
{
    // Modified index i for several blocks
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // added index for second matrix dimension
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // changed the indexing of the arrays to accessing matrix elements
    c[i][j] = a[i][j] + b[i][j];
}

int main()
{
    const int matrixSizeX = SIZE;
    const int matrixSizeY = SIZE;
    // changed into 2D arrays
    const int a[matrixSizeX][SIZE] = { {1, 2, 3}, {4, 5, 6}, {7, 8, 9 } };
    const int b[matrixSizeX][SIZE] = { {10, 20, 30}, {40, 50, 60}, {70, 80, 90 } };
    int c[matrixSizeX][SIZE] = { 0 };

    // Added variable for storing threads number
    dim3 threads(2, 2);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, (matrixSizeX * matrixSizeY), threads);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Modified result printing
    printf("{{1, 2, 3}, {4, 5, 6}, {7, 8, 9 }} + {{10, 20, 30}, {40, 50, 60}, {70, 80, 90}} = {{%d,%d,%d},{%d,%d,%d},{%d,%d,%d}}\n",
       c[0][0], c[0][1], c[0][2], c[1][0], c[1][1], c[1][2], c[2][0], c[2][1], c[2][2] );

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
// added number of threads (block size) as a parameter
// added total matrix size as a parameter for proper memory allocation
// changed 1D arrays for the 2D matrices with one dimension set - C did not allow dynamic arrays
hipError_t addWithCuda(int c[][SIZE], const int a[][SIZE], const int b[][SIZE], unsigned int size, dim3 threads)
{
    // changed the declaration of CUDA arrays to 2D matrices with one dimension size set
    int (*dev_a)[SIZE] = { 0 };
    int (*dev_b)[SIZE] = { 0 };
    int (*dev_c)[SIZE] = { 0 };
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // added the calculation of the matrix size
    const int dim = sqrt(size);
    // Added calculation of blocks number in both dimensions of the matrices
    dim3 blocks ((dim + threads.x - 1) / threads.x, (dim + threads.y - 1) / threads.y);

    //printf("%d\n", blocks);

    // Added block variable to addKernel function
    // Launch a kernel on the GPU with one thread for each element.
    // parametrized the kernel values with number and size of blocks
    addKernel<<<blocks, threads>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
